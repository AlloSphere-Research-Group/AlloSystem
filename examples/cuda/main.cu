#include <stdio.h>

#include "cuda_hello.cuh"

const int N = 16;
const int blocksize = 16;

int main(int argc, char **argv){
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);

	printf("%s", a);

	hipMalloc( (void**)&ad, csize );
	hipMalloc( (void**)&bd, isize );
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
	hipFree( ad );
	hipFree( bd );

	printf("%s\n", a);
	return EXIT_SUCCESS;
}
